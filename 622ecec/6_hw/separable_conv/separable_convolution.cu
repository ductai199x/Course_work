#include "hip/hip_runtime.h"
/* Host code that implements a  separable convolution filter of a 
 * 2D signal with a gaussian kernel.
 * 
 * Author: Naga Kandasamy
 * Date modified: May 26, 2020
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>
#include "seperable_convolution.h"

extern "C" void compute_gold(float *, float *, int, int, int);
extern "C" float *create_kernel(float, int);
void print_kernel(float *, int);
void print_matrix(float *, int, int);

/* Uncomment line below to spit out debug information */
//#define DEBUG

/* Include device code */
#include "separable_convolution_kernel.cu"

void compute_on_device(float *gpu_result, float *matrix_c,\
                   float *kernel, int num_cols,\
                   int num_rows, int half_width, int is_opt);

int main(int argc, char **argv)
{
    if (argc < 3) {
        printf("Usage: %s num-rows num-columns\n", argv[0]);
        printf("num-rows: height of the matrix\n");
        printf("num-columns: width of the matrix\n");
        exit(EXIT_FAILURE);
    }

    int num_rows = atoi(argv[1]);
    int num_cols = atoi(argv[2]);

    /* Create input matrix */
    int num_elements = num_rows * num_cols;
    printf("Creating input matrix of %d x %d\n", num_rows, num_cols);
    float *matrix_a = (float *)malloc(sizeof(float) * num_elements);
    float *matrix_c = (float *)malloc(sizeof(float) * num_elements);
	
    srand(time(NULL));
    int i;
    for (i = 0; i < num_elements; i++) {
        matrix_a[i] = rand()/(float)RAND_MAX;			 
        matrix_c[i] = matrix_a[i]; /* Copy contents of matrix_a into matrix_c */
    }
	 
	/* Create Gaussian kernel */	  
    float *gaussian_kernel = create_kernel((float)COEFF, HALF_WIDTH);	
//    print_kernel(gaussian_kernel, HALF_WIDTH);
	  
    /* Convolve matrix along rows and columns. 
       The result is stored in matrix_a, thereby overwriting the 
       original contents of matrix_a.		
     */
    printf("\nConvolving the matrix on the CPU\n");
	struct timeval start, stop;
	gettimeofday(&start, NULL);
    compute_gold(matrix_a, gaussian_kernel, num_cols,\
                  num_rows, HALF_WIDTH);
    gettimeofday(&stop, NULL);
	fprintf(stderr, "Execution time CPU GOLD = %fs\n", (float) (stop.tv_sec - start.tv_sec
				+ (stop.tv_usec - start.tv_usec) / (float) 1000000));

    float *gpu_result = (float *)malloc(sizeof(float) * num_elements);
    
    /* FIXME: Edit this function to complete the functionality on the GPU.
       The input matrix is matrix_c and the result must be stored in 
       gpu_result.
     */
    printf("\nConvolving matrix on the GPU naive\n");
    compute_on_device(gpu_result, matrix_c, gaussian_kernel, num_cols,\
                       num_rows, HALF_WIDTH, 0);
       
    printf("\nComparing CPU and GPU results\n");
    float sum_delta = 0, sum_ref = 0;
    for (i = 0; i < num_elements; i++) {
        sum_delta += fabsf(matrix_a[i] - gpu_result[i]);
        sum_ref   += fabsf(matrix_a[i]);
    }
        
    float L1norm = sum_delta / sum_ref;
    float eps = 1e-6;
    printf("L1 norm: %E\n", L1norm);
    printf((L1norm < eps) ? "TEST PASSED\n" : "TEST FAILED\n");

    // ----------------------------------------------------------------
    printf("\nConvolving matrix on the GPU optimized\n");
    compute_on_device(gpu_result, matrix_c, gaussian_kernel, num_cols,\
                       num_rows, HALF_WIDTH, 1);

    printf("\nComparing CPU and GPU results\n");
    for (i = 0; i < num_elements; i++) {
        sum_delta += fabsf(matrix_a[i] - gpu_result[i]);
        sum_ref   += fabsf(matrix_a[i]);
    }

    printf("L1 norm: %E\n", L1norm);
    printf((L1norm < eps) ? "TEST PASSED\n" : "TEST FAILED\n");

    free(matrix_a);
    free(matrix_c);
    free(gpu_result);
    free(gaussian_kernel);

    exit(EXIT_SUCCESS);
}

/* FIXME: Edit this function to compute the convolution on the device.*/
void compute_on_device(float *gpu_result, float *matrix_c,\
                   float *kernel, int num_cols,\
                   int num_rows, int half_width, int is_opt)
{
	int size = num_rows*num_cols*sizeof(float);
	int kernel_size = (half_width*2+1)*sizeof(float);

	float* kernel_dev;
	float* result_dev;
	float* c_dev;
	// allocate memory on device
	hipMalloc((void **)&result_dev, size);
	hipMalloc((void **)&c_dev, size);
	hipMalloc((void **)&kernel_dev, kernel_size);

	// copy to device
	hipMemcpy(result_dev, gpu_result, size, hipMemcpyHostToDevice);
	hipMemcpy(c_dev, matrix_c, size, hipMemcpyHostToDevice);
	if (is_opt)
		hipMemcpyToSymbol(HIP_SYMBOL(const_kernel_dev), kernel, kernel_size);
	else
		hipMemcpy(kernel_dev, kernel, kernel_size, hipMemcpyHostToDevice);

    // set up the execution grid on device
	dim3 thread_block(32, 32);
	dim3 grid((num_rows + 32 - 1) / 32, (num_cols + 32 - 1) / 32);
	printf("Creating grid of %d x %d", (num_rows + 32 - 1) / 32, (num_cols + 32 - 1) / 32);

	// timer start
	struct timeval start, stop;
	gettimeofday(&start, NULL);

	// launch kernel
	if (is_opt)
		convolve_rows_kernel_opt<<<grid, thread_block>>>(result_dev, c_dev, kernel_dev, num_rows, num_cols, half_width);
	else
		convolve_rows_kernel_naive<<<grid, thread_block>>>(result_dev, c_dev, kernel_dev, num_rows, num_cols, half_width);
	hipDeviceSynchronize();
	hipMemcpy(gpu_result, result_dev, size, hipMemcpyDeviceToHost);
	hipMemcpy(c_dev, gpu_result, size, hipMemcpyHostToDevice);

	//switch x, y of grid
//	grid.x = 1; grid.y = (num_rows + 32 - 1) / 32;
	if (is_opt)
		convolve_columns_kernel_opt<<<grid, thread_block>>>(result_dev, c_dev, kernel_dev, num_rows, num_cols, half_width);
	else
		convolve_columns_kernel_naive<<<grid, thread_block>>>(result_dev, c_dev, kernel_dev, num_rows, num_cols, half_width);
	hipDeviceSynchronize();

	// timer stop
    gettimeofday(&stop, NULL);
    if (is_opt)
		fprintf(stderr, "Execution time GPU OPT = %fs\n", (float) (stop.tv_sec - start.tv_sec
					+ (stop.tv_usec - start.tv_usec) / (float) 1000000));
    else
    	fprintf(stderr, "Execution time GPU NAIVE = %fs\n", (float) (stop.tv_sec - start.tv_sec
					+ (stop.tv_usec - start.tv_usec) / (float) 1000000));

	// copy result from device to host
	hipMemcpy(gpu_result, result_dev, size, hipMemcpyDeviceToHost);

	hipFree(kernel_dev);
	hipFree(result_dev);
	hipFree(c_dev);

    return;
}


/* Check for errors reported by the CUDA run time */
void check_for_error(char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA ERROR: %s (%s)\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    return;
} 

/* Print convolution kernel */
void print_kernel(float *kernel, int half_width)
{
    int i, j = 0;
    for (i = -half_width; i <= half_width; i++) {
        printf("%0.2f ", kernel[j]);
        j++;
    }

    printf("\n");
    return;
}

/* Print matrix */
void print_matrix(float *matrix, int num_cols, int num_rows)
{
    int i,  j;
    float element;
    for (i = 0; i < num_rows; i++) {
        for (j = 0; j < num_cols; j++){
            element = matrix[i * num_cols + j];
            printf("%0.2f ", element);
        }
        printf("\n");
    }

    return;
}

